#include "hip/hip_runtime.h"
#include "utility/utils.h"


gridblock gb1d(const unsigned n, const unsigned block_size, const bool is_warp_agg, int MINI_WARP_SIZE){
  gridblock gb;

  int n_ = n;

  if(is_warp_agg)
    n_ *= MINI_WARP_SIZE;

  dim3 block (block_size);
  dim3 grid ( ceil( (double) n_ / (double) block.x));

  gb.b = block;
  gb.g = grid;

  //printf("%d %d\n\n", gb.g.x, gb.b.x);

  return gb;
}


// cuSPARSE API errors
const char* cusparseGetStatusString(hipsparseStatus_t error){
    switch (error)
    {
        case HIPSPARSE_STATUS_SUCCESS:                  return "HIPSPARSE_STATUS_SUCCESS";
        case HIPSPARSE_STATUS_NOT_INITIALIZED:          return "HIPSPARSE_STATUS_NOT_INITIALIZED";
        case HIPSPARSE_STATUS_ALLOC_FAILED:             return "HIPSPARSE_STATUS_ALLOC_FAILED";
        case HIPSPARSE_STATUS_INVALID_VALUE:            return "HIPSPARSE_STATUS_INVALID_VALUE";
        case HIPSPARSE_STATUS_ARCH_MISMATCH:            return "HIPSPARSE_STATUS_ARCH_MISMATCH";
        case HIPSPARSE_STATUS_MAPPING_ERROR:            return "HIPSPARSE_STATUS_MAPPING_ERROR";
        case HIPSPARSE_STATUS_EXECUTION_FAILED:         return "HIPSPARSE_STATUS_EXECUTION_FAILED";
        case HIPSPARSE_STATUS_INTERNAL_ERROR:           return "HIPSPARSE_STATUS_INTERNAL_ERROR";
        case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";
    }
    return "<unknown>";
}



const char* cublasGetStatusString(hipblasStatus_t status) {
  switch(status) {
    case HIPBLAS_STATUS_SUCCESS:           return "HIPBLAS_STATUS_SUCCESS";
    case HIPBLAS_STATUS_NOT_INITIALIZED:   return "HIPBLAS_STATUS_NOT_INITIALIZED";
    case HIPBLAS_STATUS_ALLOC_FAILED:      return "HIPBLAS_STATUS_ALLOC_FAILED";
    case HIPBLAS_STATUS_INVALID_VALUE:     return "HIPBLAS_STATUS_INVALID_VALUE";
    case HIPBLAS_STATUS_ARCH_MISMATCH:     return "HIPBLAS_STATUS_ARCH_MISMATCH";
    case HIPBLAS_STATUS_MAPPING_ERROR:     return "HIPBLAS_STATUS_MAPPING_ERROR";
    case HIPBLAS_STATUS_EXECUTION_FAILED:  return "HIPBLAS_STATUS_EXECUTION_FAILED";
    case HIPBLAS_STATUS_INTERNAL_ERROR:    return "HIPBLAS_STATUS_INTERNAL_ERROR";
    case HIPBLAS_STATUS_NOT_SUPPORTED:     return "HIPBLAS_STATUS_NOT_SUPPORTED";
    case HIPBLAS_STATUS_UNKNOWN:     return "HIPBLAS_STATUS_UNKNOWN";
  }
  return "CUBLAS_STATUS_UNKNOWN_ERROR";
}

void CHECK_CUBLAS(hipblasStatus_t err){
  const char *err_str = cublasGetStatusString(err);
  if(err != HIPBLAS_STATUS_SUCCESS){
    printf("[ERROR CUBLAS] :\n\t%s\n", err_str);
    exit(1);
  }
}


//##############################################################################

namespace Parallel{
  template <typename T>
  T* max(T *a, int n, bool host_result, hipStream_t stream){

    void     *d_temp_storage = NULL;
    size_t   temp_storage_bytes = 0;

    T *max = NULL;
    hipError_t err;
    err = hipMalloc((void**)&max, sizeof(T));
    CHECK_DEVICE(err);

    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, a, max, n, stream);
    // Allocate temporary storage
    err = hipMalloc(&d_temp_storage, temp_storage_bytes);
    CHECK_DEVICE(err);
    // Run max-reduction
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, a, max, n, stream);

    err = hipFree(d_temp_storage);
    CHECK_DEVICE(err);

    if(host_result){
      T *max_host = (T*) malloc(sizeof(T));
      CHECK_HOST(max_host);

      CHECK_DEVICE( hipMemcpy(max_host, max, sizeof(T), hipMemcpyDeviceToHost) );
      CHECK_DEVICE( hipFree(max) );
      return max_host;
    }

    return max;
  }

  template <typename T>
  T* cumsum(T *a, int n, bool host_result, hipStream_t stream){

    T *out = NULL;
    CHECK_DEVICE( hipMalloc((void**)&out, n * sizeof(T)) );

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, a, out, n);
    // Allocate temporary storage for inclusive prefix sum
    hipError_t err = hipMalloc(&d_temp_storage, temp_storage_bytes);
    CHECK_DEVICE(err);
    // Run inclusive prefix sum
    hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, a, out, n);

    CHECK_DEVICE( hipFree(d_temp_storage) );


    if(host_result){
      T *out_host = (T*) malloc(sizeof(T));
      CHECK_HOST(out_host);

      CHECK_DEVICE( hipMemcpy(out_host, out, n * sizeof(T), hipMemcpyDeviceToHost) );
      CHECK_DEVICE( hipFree(out) );
      return out_host;
    }

    return out;
  }
}

//##############################################################################

namespace TIME{

  int timer_index;
  int n;
  hipEvent_t *starts, *stops;

  void init(){
    TIME::timer_index = 0;
    TIME::n = 0;
    TIME::starts = NULL;
    TIME::stops = NULL;
  }

  void addTimer(){
    TIME::starts = (hipEvent_t*) realloc(TIME::starts, sizeof(hipEvent_t) * TIME::n);
    CHECK_HOST(TIME::starts);
    TIME::stops = (hipEvent_t*) realloc(TIME::stops, sizeof(hipEvent_t) * TIME::n);
    CHECK_HOST(TIME::stops);
    hipEventCreate(&TIME::starts[TIME::n-1]);
    hipEventCreate(&TIME::stops[TIME::n-1]);
  }

  void start(){
    if(TIME::timer_index == TIME::n){
      TIME::n++;
      TIME::addTimer();
    }
    hipEventRecord(TIME::starts[TIME::timer_index]);
    TIME::timer_index++;
  }

  float stop(){
    float milliseconds = 0.;
    hipEvent_t start_ = TIME::starts[TIME::timer_index-1];
    hipEvent_t stop_ = TIME::stops[TIME::timer_index-1];

    hipEventRecord(stop_);
    hipEventSynchronize(stop_);
    hipEventElapsedTime(&milliseconds, start_, stop_);
    TIME::timer_index--;
    return milliseconds;
  }

  void free(){
    for(int i=0; i<TIME::n; i++){
      hipEventDestroy( TIME::starts[i]);
      hipEventDestroy( TIME::stops[i]);
    }
    std::free( TIME::starts);
    std::free( TIME::stops);
  }
}
