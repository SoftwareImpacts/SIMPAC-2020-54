#define PRG_SEED 1

struct handles{
  hipStream_t stream1;
  hipsparseHandle_t cusparse_h0, cusparse_h1;
  hipblasHandle_t hipblas.h;
  hipsolverSpHandle_t cusolver_h;
  hiprandGenerator_t uniformRNG;
};

namespace Handles{
  handles* init(){

    handles *h = (handles*) malloc(sizeof(handles));
    CHECK_HOST(h);

    CHECK_CUSPARSE( hipsparseCreate(&(h->cusparse_h0)) );
    CHECK_CUSPARSE( hipsparseCreate(&(h->cusparse_h1)) );

    CHECK_CUBLAS( hipblasCreate(&(h->cublas_h)) );

    CHECK_DEVICE( hipStreamCreate(&(h->stream1)) );

    CHECK_CUSPARSE( hipsparseSetStream(h->cusparse_h1, h->stream1) );

    CHECK_CUSOLVER( hipsolverSpCreate(&(h->cusolver_h)) );

    hiprandCreateGenerator(&h->uniformRNG, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(h->uniformRNG, PRG_SEED);

    return h;
  }

  void free(handles *h){
    CHECK_CUSPARSE( hipsparseDestroy(h->cusparse_h0) );
    CHECK_CUSPARSE( hipsparseDestroy(h->cusparse_h1) );

    CHECK_CUBLAS( hipblasDestroy(h->cublas_h) );

    CHECK_DEVICE( hipStreamDestroy(h->stream1) );

    CHECK_CUSOLVER( hipsolverSpDestroy(h->cusolver_h) );

    //hiprandDestroyGenerator(h->uniformRNG);

    std::free(h);
  }
}
